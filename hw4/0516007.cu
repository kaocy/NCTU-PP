/**********************************************************************
 * DESCRIPTION:
 *   Parallel Concurrent Wave Equation - C with CUDA Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints; 	     		    /* total points along string */   
float *values;                  /* values in the end, in host */
float *doldval;                 /* values at time (t-dt), in device */
float *dnewval;                 /* values at time (t), in device */

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void) {
   char tchar[20];

   /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ void init_line(float *doldval, float *dnewval, int tpoints) {
    int index = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (index > tpoints)    return ;

    /* Calculate initial values based on sine curve */
    float fac = 2.0 * PI;
    float x = (float)(index - 1) / (tpoints - 1);
    doldval[index] = dnewval[index] = sin(fac * x);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
 __global__ void update(float *doldval, float *dnewval, int tpoints, int nsteps) {
    int index = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (index > tpoints)    return ;

    /* Update values for each time step */
    for (int i = 1; i <= nsteps; i++) {
        float value;
        /* global endpoints */
        if ((index == 1) || (index  == tpoints))
            value = 0.0;
        else
            value = (2.0 * dnewval[index]) - doldval[index] + (-0.18 * dnewval[index]);

        /* Update old values with new values */
        doldval[index] = dnewval[index];
        dnewval[index] = value;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal() {
    for (int i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i % 10 == 0)  printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[]) {
	sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
    check_param();

    const int block_size = 256;
    int block_num = tpoints / block_size + ((tpoints % block_size) > 0);
    const int array_size = (tpoints + 2) * sizeof(float);

    hipMalloc(&doldval, array_size);
    hipMalloc(&dnewval, array_size);

	printf("Initializing points on the line...\n");
    init_line<<<block_num, block_size>>>(doldval, dnewval, tpoints);

	printf("Updating all points for all time steps...\n");
    update<<<block_num, block_size>>>(doldval, dnewval, tpoints, nsteps);

    values = (float*) malloc(array_size);
    hipMemcpy(values, dnewval, array_size, hipMemcpyDeviceToHost);

	printf("Printing final results...\n");
	printfinal();
    printf("\nDone.\n\n");
    
    free(values);
    hipFree(doldval);
    hipFree(dnewval);
	
	return 0;
}